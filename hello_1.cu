/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc hello.cu -o hello -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  printf("Hello World by Christopher García (20541)\n");

  if (tid == 1023) {
    printf("Thread 1023, Christopher García (20541)\n");
  }
}

int main()
{
  hello<<<2,1023>>>();
  hipDeviceReset();
  return 0;
}
